#include "kernels_list.hpp"
#include "gj-reference.hpp"
#include "gj-flawed.hpp"
#include "cuda-utils.hpp"


void check_cuda_error(hipError_t error_code,const char* file, int line){
    if(error_code != hipSuccess){
        std::string msg = std::string("CUDA Error : ") + hipGetErrorString(error_code) + std::string(" in : ") + file + std::string(" line ") + std::to_string(line);
        throw std::runtime_error(msg);
    }
}

void retreive_kernels(){
    hipSetDevice(0);
    KernelsManager* km = KernelsManager::instance();
    km->registerKernel("BS", std::make_shared<ReferenceGaussJordan>());
    km->registerKernel("OC",std::make_shared<OCGaussJordan>());
    km->registerKernel("RC",std::make_shared<RCGaussJordan>());
    km->registerKernel("DA",std::make_shared<DAGaussJordan>());
}



CudaProfiling::CudaProfiling(){
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
};

CudaProfiling::~CudaProfiling(){
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));
};

void CudaProfiling::begin(){
    CHECK_CUDA(hipEventRecord(start));
}

ExecutionStats CudaProfiling::end(){
    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));

    ExecutionStats stats;
    CHECK_CUDA(hipEventElapsedTime(&stats.elapsed, start, stop));
    return stats;
}