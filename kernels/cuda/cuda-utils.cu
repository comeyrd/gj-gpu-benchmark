#include "gj-reference.hpp"
#include "gj-flawed.hpp"
#include "cuda-utils.hpp"

void check_cuda_error(hipError_t error_code,const char* file, int line){
    if(error_code != hipSuccess){
        std::string msg = std::string("CUDA Error : ") + hipGetErrorString(error_code) + std::string(" in : ") + file + std::string(" line ") + std::to_string(line);
        throw std::runtime_error(msg);
    }
}

void setup_gpu(){
    hipSetDevice(0);
}

void reset_state(){
    hipDeviceReset();
}

CudaProfiling::CudaProfiling(){
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
};

CudaProfiling::~CudaProfiling(){
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));
};

void CudaProfiling::begin(){
    CHECK_CUDA(hipEventRecord(start));
}

ExecutionStats CudaProfiling::end(){
    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));

    ExecutionStats stats;
    CHECK_CUDA(hipEventElapsedTime(&stats.elapsed, start, stop));
    return stats;
}