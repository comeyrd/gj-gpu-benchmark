#include "hip/hip_runtime.h"
#include "gj-rm.hpp"
#include "cuda-utils.hpp"
//Array with a fixed size, unused memory

__global__ void rm_fixRow(double *matrix, int size,int rowId){
    __shared__ double Ri[1024];//bug
    __shared__ double Aii;

    int colId = threadIdx.x;
    Ri[colId] = matrix[size*rowId + colId];
    if(colId == rowId)
        Aii = Ri[rowId];
    __syncthreads();
    Ri[colId] = Ri[colId] / Aii;
    matrix[size*rowId+colId] = Ri[colId];
}

__global__ void rm_fixColumn(double *matrix, int size, int colId){
    int col_x = threadIdx.x;
    int row_x = blockIdx.x;
    __shared__ double ratio;
    if(row_x!=colId && matrix[row_x*size + colId] != 0){
        if(col_x == 0)
            ratio = matrix[row_x*size + colId] / matrix[colId*size + colId];
        __syncthreads();
        double val = matrix[row_x*size + col_x] - ratio * matrix[colId*size+col_x];
        matrix[row_x*size +col_x] = val;
    }
}

ExecutionStats rm_kernel(GJ_Utils::GJ_Matrix* m,GJ_Utils::S_Matrix* o){
    CudaProfiling prof;
    double* matrix;
    CHECK_CUDA(hipMalloc(&matrix,m->cols*m->rows*sizeof(double)));
    CHECK_CUDA(hipMemcpy(matrix,m->data.get(),m->cols*m->rows*sizeof(double),hipMemcpyHostToDevice));
    prof.begin();
    for(int l=0;l<m->rows;l++){
        rm_fixRow<<<1,m->cols>>>(matrix,m->cols,l);
        CHECK_CUDA(hipGetLastError());
        rm_fixColumn<<<m->rows,m->cols>>>(matrix,m->cols,l);
        CHECK_CUDA(hipGetLastError());
    }
    ExecutionStats stats = prof.end();

    GJ_Utils::GJ_Matrix out_gj = GJ_Utils::GJ_Matrix(m->rows);

    CHECK_CUDA(hipMemcpy(out_gj.data.get(),matrix,out_gj.cols*out_gj.rows*sizeof(double),hipMemcpyDeviceToHost));
    CHECK_CUDA(hipFree(matrix));

    GJ_Utils::S_Matrix s = out_gj.get_right_side();
    *o = out_gj.get_right_side();

    return stats;
};
REGISTER_KERNEL(RMGaussJordan)
