#include "hip/hip_runtime.h"
#include "gj-rc.hpp"
#include "cuda-utils.hpp"
//Array with a fixed size, unused memory

__global__ void rc_fixRow(double *matrix, int size,int rowId){
    __shared__ double Ri[1024];//bug
    __shared__ double Aii;

    int colId = threadIdx.x;
    Ri[colId] = matrix[size*rowId + colId];
    if(colId == rowId)
        Aii = Ri[rowId];
    __syncthreads();
    Ri[colId] = Ri[colId] / Aii;
    matrix[size*rowId+colId] = Ri[colId];
}

__global__ void rc_fixColumn(double *matrix, int size, int colId){
    int col_x = threadIdx.x;
    int row_x = blockIdx.x;
    __shared__ double ratio;
    if(row_x!=colId && matrix[row_x*size + colId] != 0){
        if(col_x == 0)
            ratio = matrix[row_x*size + colId] / matrix[colId*size + colId];
        __syncthreads();
        double val = matrix[row_x*size + col_x] - ratio * matrix[colId*size+col_x];
        matrix[row_x*size +col_x] = val;
    }
}

ExecutionStats rc_kernel(GJ_Utils::GJ_Matrix* m,GJ_Utils::S_Matrix* o){
    CudaProfiling prof;
    double* matrix;
    CHECK_CUDA(hipMalloc(&matrix,m->cols*m->rows*sizeof(double)));
    CHECK_CUDA(hipMemcpy(matrix,m->data,m->cols*m->rows*sizeof(double),hipMemcpyHostToDevice));
    prof.begin();
    for(int l=0;l<m->rows;l++){
        rc_fixRow<<<1,m->cols>>>(matrix,m->cols,l);
        CHECK_CUDA(hipGetLastError());
        rc_fixColumn<<<m->rows,m->cols>>>(matrix,m->cols,l);
        CHECK_CUDA(hipGetLastError());
    }
    ExecutionStats stats = prof.end();

    GJ_Utils::GJ_Matrix out_gj = GJ_Utils::GJ_Matrix(m->rows);

    CHECK_CUDA(hipMemcpy(out_gj.data,matrix,out_gj.cols*out_gj.rows*sizeof(double),hipMemcpyDeviceToHost));
    CHECK_CUDA(hipFree(matrix));

    GJ_Utils::S_Matrix s = out_gj.get_right_side();

    double* inner_out =  new double[s.rows * s.cols]();
    memcpy(inner_out,s.data,s.rows*s.cols*sizeof(double));
    bool o_owns_mem = true;
    o->update_memory(inner_out,o_owns_mem,s.rows,s.cols);

    return stats;
};