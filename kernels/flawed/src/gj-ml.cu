#include "hip/hip_runtime.h"
#include "gj-ml.hpp"
#include "cuda-utils.hpp"
//Leaking Memory
__global__ void ml_fixRow(double *matrix, int size,int rowId){
    extern __shared__ double Ri[];
    __shared__ double Aii;

    int colId = threadIdx.x;
    Ri[colId] = matrix[size*rowId + colId];
    if(colId == rowId)
        Aii = Ri[rowId];
    __syncthreads();
    Ri[colId] = Ri[colId]/Aii;
    matrix[size*rowId+colId] = Ri[colId];
}

__global__ void ml_fixColumn(double *matrix, int size, int colId){
    int col_x = threadIdx.x;
    int row_x = blockIdx.x;
    __shared__ double ratio;
    if(row_x!=colId && matrix[row_x*size + colId] != 0){
        if(col_x == 0)
            ratio = matrix[row_x*size + colId] / matrix[colId*size + colId];
        __syncthreads();
        double val = matrix[row_x*size + col_x] - ratio * matrix[colId*size+col_x];
        matrix[row_x*size +col_x] = val;
    }
}

ExecutionStats ml_kernel(GJ_Utils::GJ_Matrix* m,GJ_Utils::S_Matrix* o){
    CudaProfiling prof;

    double* matrix;
    CHECK_CUDA(hipMalloc(&matrix,m->cols*m->rows*sizeof(double)));
    CHECK_CUDA(hipMemcpy(matrix,m->data,m->cols*m->rows*sizeof(double),hipMemcpyHostToDevice));
    prof.begin();
    for(int l=0;l<m->rows;l++){
        ml_fixRow<<<1,m->cols,m->cols*sizeof(double)>>>(matrix,m->cols,l);
        CHECK_CUDA(hipGetLastError());
        ml_fixColumn<<<m->rows,m->cols>>>(matrix,m->cols,l);
        CHECK_CUDA(hipGetLastError());
    }
    ExecutionStats stats = prof.end();

    GJ_Utils::GJ_Matrix out_gj = GJ_Utils::GJ_Matrix(m->rows);

    CHECK_CUDA(hipMemcpy(out_gj.data,matrix,out_gj.cols*out_gj.rows*sizeof(double),hipMemcpyDeviceToHost));

    //CHECK_CUDA(hipFree(matrix));//BUG

    GJ_Utils::S_Matrix s = out_gj.get_right_side();

    double* inner_out =  new double[s.rows * s.cols]();
    memcpy(inner_out,s.data,s.rows*s.cols*sizeof(double));
    bool o_owns_mem = true;
    o->update_memory(inner_out,o_owns_mem,s.rows,s.cols);

    return stats;
};