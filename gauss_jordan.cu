#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <iomanip>

void init_square_matrix(float* matrix, int size);
void display_square_matrix(float* matrix, int size);
void init_square_matrix_wId(float* matrix, int size);
void display_square_matrix_wId(float* matrix, int size);

__global__ void fixRow(float *matrix, int size,int rowId){
    int colId = threadIdx.x;
    matrix[size*rowId + colId] = matrix[size*rowId + colId] / matrix[size*rowId + rowId];
}

__global__ void fixColumn(float *matrix, int wideness,int current_col){
    int i = threadIdx.x; // What row
    int j = blockIdx.x; //What column
    matrix[i*wideness + j] = matrix[i*wideness + j] - (matrix[i*widenesss + current_col] / matrix[current_col * wideness + current_col]) * matrix[current_col*wideness + j];
}

int main(int argc, char** argv){
    std::cout << "Gauss Jordan on GPU" << std::endl;
    int N = 3;
    float *matrix;
    hipMallocManaged(&matrix, N*(N+N)*sizeof(float));
    init_square_matrix_wId(matrix,N);
    int row = N*2;
    int col = N;

    fixRow<<<1,row>>>(matrix,row,0);
    hipDeviceSynchronize();
    fixColumn<<<row,col>>>(matrix,row,0);
    hipDeviceSynchronize();
    display_square_matrix_wId(matrix,N);
    return 0;
}


void init_square_matrix(float* matrix, int size){
    for(int i=0;i<size*size;i++){
        matrix[i] = i + 4;
    }   
}

void init_square_matrix_wId(float* matrix, int size){
    for(int i=0;i<size;i++){
        for(int k=0;k<size;k++){
            matrix[i*size*2 + k] = i + 4;
        }
    }   
    for(int s=0;s<size;s++){
        for(int m=0;m<size;m++){
            if(s == m)
                matrix[s*size*2+ m + size] = 1;
            else
                matrix[s*size*2+ m + size ] = 0;
        }
    }
}


void display_square_matrix(float* matrix, int size){
    for(int i=0;i<size*size;i++){
        if(i == 0){
            std::cout << "[ ";
        }
        std::cout << std::setw(4) << std::setfill(' ') << std::setprecision(3)<< matrix[i] << " ";
        if(i==size*size-1){
            std::cout << "]" << std::endl;
        }
        else if((i+1)%size==0 && i!=0){
            std::cout << std::endl<< "  ";
        }
    } 

} 

void display_square_matrix_wId(float* matrix, int size){
    for(int i=0;i<size*(size*2);i++){
        if(i == 0){
            std::cout << "[ ";
        }
        std::cout << std::setw(4) << std::setfill(' ') << std::setprecision(3)<< matrix[i] << " ";
        
        if(i==(size*2)*size-1){
            std::cout << "]" << std::endl;
        }
        else if((i+1)%(size*2)==0 && i!=0){
            std::cout << std::endl<< "  ";
        }else if((i+1)%(size)==0 && i!=0){
            std::cout << " | ";
        }
    } 

} 