#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <iomanip>
#include <stdio.h>
#include "gj_utils.hpp"
#include <cmath>



__global__ void fixRow(double *matrix, int size,int rowId){
    __shared__ double Ri[512];
    __shared__ double Aii;

    int colId = threadIdx.x;
    Ri[colId] = matrix[size*rowId + colId];
    Aii = matrix[size*rowId + rowId];
    __syncthreads();//Block synchronisation barrier
    Ri[colId] = Ri[colId]/Aii;
    matrix[size*rowId+colId] = Ri[colId];
}

__global__ void fixColumn(double *matrix, int size, int colId){
    int i = threadIdx.x;
    int j = blockIdx.x;
    __shared__ double col[512];
    __shared__ double colj[512];
    __shared__ double AColIdj;
    col[i] = matrix[i * size + colId];
    if(col[i]!=0){
        colj[i] = matrix[i * size + colId];
        AColIdj = matrix[colId * size + j];
        if (i!= colId){
            colj[i] = colj[i] - AColIdj * col[i];
        }
        matrix[i * size + j] = colj[i];
    }
}

__global__ void myfixColumn(double *matrix, int size, int colId){
    int col_x = threadIdx.x;
    int row_x = blockIdx.x;
    __shared__ double ratio;
    if(row_x!=colId && matrix[row_x*size + colId] != 0){
        ratio = matrix[row_x*size + colId] / matrix[colId*size + colId];
        double val = matrix[row_x*size + col_x] - ratio * matrix[colId*size+col_x];
        matrix[row_x*size +col_x] = val;
    }
}


__global__ void perform_swap(double *matrix, int size, int colId,int swapId){
    int col_x = threadIdx.x;
    double toswap = matrix[colId*size+col_x];
    matrix[colId*size+col_x] = matrix[swapId*size+col_x];
    matrix[swapId*size+col_x] = toswap;
}

int main(int argc, char** argv){
    std::cout << "Gauss Jordan on GPU" << std::endl;
    int N = 5;
    bool with_swap = false;
    bool debug = false;
    if(argc == 4){
        N = std::atoi(argv[1]);
        if((strcmp(argv[2], "1") == 0)){
            with_swap = true;
        }
        if((strcmp(argv[3], "1") == 0)){
            debug = true;
        }
    }
    GJ_Utils::S_Matrix m1 = GJ_Utils::S_Matrix(N);
    m1.fill_random_U();
    GJ_Utils::S_Matrix m2 = GJ_Utils::S_Matrix(N);
    m2.fill_random_L();
    GJ_Utils::S_Matrix m3 = m2.times(&m1);
    if(debug){
        std::cout<<"Base matrix : "<<std::endl;
        m3.print();
    }
    double *matrix;
    hipMallocManaged(&matrix, N*(N+N)*sizeof(double));

    GJ_Utils::GJ_Matrix gjm1 =  GJ_Utils::GJ_Matrix(matrix,&m3);
    //gjm1.print();

    
    int col = N*2;
    int row = N;
    for(int l=0;l<row;l++){
        if(with_swap){
            double max = std::fabs(matrix[l*col+l]);
            int swapId;
            for(int i=l+1;i<row;i++){
            if(std::fabs(matrix[i*col+l]) > max){
                max = std::fabs(matrix[i*col+l]);
                swapId = i;
            }
            }
            if(max>std::fabs(matrix[l*col+l])){
                if(debug)
                    std::cout <<"Swapping row "<< l<< " and "<<swapId <<std::endl;
                perform_swap<<<1,col>>>(matrix,col,l,swapId);
                hipDeviceSynchronize();
            }
        }
        fixRow<<<1,col>>>(matrix,col,l);
        hipDeviceSynchronize();
        myfixColumn<<<row,col>>>(matrix,col,l);
        hipDeviceSynchronize();
        if(debug){
            std::cout<<"Row "<<l<<std::endl;
            gjm1.print();
        }
    }
    if(debug){
        std::cout<<"Matrix after Gj: "<<std::endl;
        gjm1.print();
    }
    GJ_Utils::S_Matrix ls = gjm1.get_right_side();
    auto [inv,max_error] = ls.is_inverse(&m3);
    std::cout << "My method is inverse : " << inv << " With max error : "<< max_error <<std::endl;
    if(!inv && debug){
        GJ_Utils::S_Matrix invt = ls.times(&m3);
        invt.print();
    }
    
    return 0;
} 
